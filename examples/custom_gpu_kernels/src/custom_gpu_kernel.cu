
#include <hip/hip_runtime.h>
#include <cstdint>
#include <cstdio>
__global__ void my_gpu_kernel(double *dst, const double *src, size_t number_of_elements) {
        uint64_t idx = (uint64_t)(blockIdx.x)*(uint64_t)(blockDim.x) + threadIdx.x;
        if (idx < number_of_elements)
                dst[idx] = 1.0/((src[idx]+1)*(src[idx]+1));
}

#define max(a,b) ((a)> (b) ? (a) : (b))

extern "C" void my_gpu_callback(double *dst, double *src, size_t number_of_elements, hipStream_t *stream) {
        const size_t block_size = 256;
        const size_t number_of_blocks = max(number_of_elements/block_size,1);
        if (stream) {
                my_gpu_kernel<<<number_of_blocks, block_size, 0, *stream>>>(dst, src, number_of_elements);
        } else {
                my_gpu_kernel<<<number_of_blocks, block_size>>>(dst, src, number_of_elements);
        }
}
